//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// cwtMean_initialize.cu
//
// Code generation for function 'cwtMean_initialize'
//

// Include files
#include "cwtMean_initialize.h"
#include "_coder_cwtMean_mex.h"
#include "cwt.h"
#include "cwtMean_data.h"
#include "rt_nonfinite.h"

// Function Declarations
static void cwtMean_once();

// Function Definitions
static void cwtMean_once()
{
  mex_InitInfAndNan();
  psidft_not_empty_init();
  hipMalloc(&dv1_gpu_clone, sizeof(real_T[10001]));
  hipMalloc(&dv_gpu_clone, sizeof(real_T[95]));
}

void cwtMean_initialize()
{
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal,
                          (const char_T *)"distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal, (const char_T *)"wavelet_toolbox",
                          2);
  if (emlrtFirstTimeR2012b(emlrtRootTLSGlobal)) {
    cwtMean_once();
  }
  hipGetLastError();
}

// End of code generation (cwtMean_initialize.cu)
