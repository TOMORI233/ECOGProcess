//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// cwtMean_terminate.cu
//
// Code generation for function 'cwtMean_terminate'
//

// Include files
#include "cwtMean_terminate.h"
#include "_coder_cwtMean_mex.h"
#include "cwtMean_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void cwtMean_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
  hipFree(*dv1_gpu_clone);
  hipFree(*dv_gpu_clone);
}

void cwtMean_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (cwtMean_terminate.cu)
