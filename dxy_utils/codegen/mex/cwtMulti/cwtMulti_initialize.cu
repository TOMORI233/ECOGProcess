//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// cwtMulti_initialize.cu
//
// Code generation for function 'cwtMulti_initialize'
//

// Include files
#include "cwtMulti_initialize.h"
#include "_coder_cwtMulti_mex.h"
#include "cwt.h"
#include "cwtMulti_data.h"
#include "rt_nonfinite.h"

// Function Declarations
static void cwtMulti_once();

// Function Definitions
static void cwtMulti_once()
{
  mex_InitInfAndNan();
  psidft_not_empty_init();
  hipMalloc(&dv1_gpu_clone, sizeof(real_T[10001]));
  hipMalloc(&dv_gpu_clone, sizeof(real_T[95]));
}

void cwtMulti_initialize()
{
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal,
                          (const char_T *)"distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal, (const char_T *)"wavelet_toolbox",
                          2);
  if (emlrtFirstTimeR2012b(emlrtRootTLSGlobal)) {
    cwtMulti_once();
  }
  hipGetLastError();
}

// End of code generation (cwtMulti_initialize.cu)
