//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// cwtMulti_terminate.cu
//
// Code generation for function 'cwtMulti_terminate'
//

// Include files
#include "cwtMulti_terminate.h"
#include "_coder_cwtMulti_mex.h"
#include "cwtMulti_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void cwtMulti_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
  hipFree(*dv1_gpu_clone);
  hipFree(*dv_gpu_clone);
}

void cwtMulti_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (cwtMulti_terminate.cu)
